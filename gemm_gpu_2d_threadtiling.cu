#include "hip/hip_runtime.h"
#pragma once

#include <cassert>
#include "gemm_gpu_2d_threadtiling.h"


template <const int BM, const int BN, const int BK, const int TM, const int TN>
__global__ void gemm_gpu_2d_threadtiling_kernel(
    const int m, 
    const int n, 
    const int k,
    float *A,
    float alpha,
    float *B,
    float beta,
    float *C
){
    //share memory.
    __shared__ float shareA[BM*BK];
    __shared__ float shareB[BK*BN];

    // use register to store result.
    float threadResults[TM*TN] = {0.0};
    float regA[TM] = {0.0};
    float regB[TN] = {0.0};
   
   
    // calculating the indices that this thread will be responsible for the result C.
    const int threadRow = (threadIdx.x*TN / BN)*TM;
    const int threadCol = threadIdx.x*TN % BN;

    A += blockIdx.x * BM * k;
    B += blockIdx.y * BN;
    C += blockIdx.x * BM * n + blockIdx.y * BN;

    int threadNumsPerBlock = BM * BN / (TM*TN);
    int strideA = threadNumsPerBlock/BK;
    int strideB = threadNumsPerBlock/BN;

    // calculating the indices that this thread will load into SMEM
    const uint innerRowA = threadIdx.x / BK; // [0,1,2,...,BM]
    const uint innerColA = threadIdx.x % BK; 
    const uint innerRowB = threadIdx.x / BN; // [0,1,2,...,BK]
    const uint innerColB = threadIdx.x % BN;

    for(uint tile_idx = 0; tile_idx < k; tile_idx += BK){
        for(uint loadoffset = 0; loadoffset < BM; loadoffset+=strideA){
            shareA[(innerRowA+loadoffset)*BK+innerColA] = A[(innerRowA+loadoffset)*k+innerColA]; 
        }
        for(uint loadoffset = 0; loadoffset < BK; loadoffset+=strideB){
            shareB[(innerRowB+loadoffset)*BN+innerColB] = B[(innerRowB+loadoffset)*n+innerColB];
        }
        __syncthreads();

        A += BK;
        B += BK*n; 
        
        for(uint dotIdx = 0; dotIdx < BK; dotIdx++){                             
            // load into register.
            for(uint i=0; i<TM; i++){
                regA[i] = shareA[(threadRow+i)*BK+dotIdx];
            }
            for(uint i=0; i<TN; i++){
                regB[i] = shareB[dotIdx*BN+threadCol+i];
            }
            for(uint resIdxM=0; resIdxM < TM; resIdxM++){
                for(uint resIdxN=0; resIdxN<TN; resIdxN++){
                    threadResults[resIdxM*TN+resIdxN] += regA[resIdxM]*regB[resIdxN];
                }
            }
        }
        __syncthreads();
    }
    for(uint resIdxM=0; resIdxM < TM; resIdxM++){
        for(uint resIdxN=0; resIdxN<TN; resIdxN++){
            C[(threadRow+resIdxM)*n+threadCol+resIdxN] = threadResults[resIdxM*TN+resIdxN];
            // float4 tmp = reinterpret_cast<float4 *>(
            //     &C[(threadRow + resIdxM) * n + threadCol + resIdxN])[0];
            // tmp.x = threadResults[resIdxM*TN + resIdxN];
            // tmp.y = threadResults[resIdxM*TN + resIdxN+1];
            // tmp.z = threadResults[resIdxM*TN + resIdxN+2];
            // tmp.w = threadResults[resIdxM*TN + resIdxN+3];
            // reinterpret_cast<float4 *>(
            //     &C[(threadRow + resIdxM) * m + threadCol + resIdxN])[0] = tmp;
        }
    }

}

void gemm_gpu_2d_threadtiling(
    const int m, 
    const int n, 
    const int k,
    float *A,
    float alpha,
    float *B,
    float beta,
    float *C,
    hipblasHandle_t handle
){
    const int BM = 128;
    const int BN = 128;
    const int BK = 8;
    const int TM = 8;
    const int TN = 4;
    // const int BM = 64;
    // const int BN = 64;
    // const int BK = 16;
    // const int TM = 8;
    // const int TN = 4;

    dim3 grid_dim = dim3(ceil(m/BM), ceil(n/BN));
    dim3 block_dim = dim3(BN*BM/(TM*TN));
    gemm_gpu_2d_threadtiling_kernel<BM, BN, BK, TM, TN><<<grid_dim, block_dim>>>(m,n,k,A,alpha,B,beta,C);
}