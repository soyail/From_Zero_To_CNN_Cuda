#include "gemm_gpu_cublas.h"

void gemm_gpu_cublas(
    const int m, 
    const int n, 
    const int k,
    float *A,
    float alpha,
    float *B,
    float beta,
    float *C,
    hipblasHandle_t handle
    ){
    hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, m, k, &alpha, B,
                n, A, k, &beta, C, n);
}
