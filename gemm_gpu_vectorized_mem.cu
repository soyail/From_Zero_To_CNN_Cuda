#include "hip/hip_runtime.h"
#pragma once


#include "gemm_gpu_vectorized_mem.h"


template <const int BM, const int BN, const int BK, const int TM, const int TN>
__global__ void gemm_gpu_vectorized_mem_kernel(
    const int m, 
    const int n, 
    const int k,
    float *A,
    float alpha,
    float *B,
    float beta,
    float *C
){
    //share memory.
    __shared__ float shareA[BM*BK];
    __shared__ float shareB[BK*BN];

    // use register to store result.
    float threadResults[TM*TN] = {0.0};
    float regA[TM] = {0.0};
    float regB[TN] = {0.0};
    
    // calculating the indices that this thread will be responsible for the result C.
    const int threadRow = (threadIdx.x*TN / BN)*TM;
    const int threadCol = threadIdx.x*TN % BN;

    A += blockIdx.x * BM * k;
    B += blockIdx.y * BN;
    C += blockIdx.x * BM * n + blockIdx.y * BN;

    // calculating the indices that this thread will load into SMEM
    const uint innerRowA = threadIdx.x / (BK/4); 
    const uint innerColA = threadIdx.x % (BK/4); 
    const uint innerRowB = threadIdx.x / (BN/4); 
    const uint innerColB = threadIdx.x % (BN/4);

    for(uint tile_idx = 0; tile_idx < k; tile_idx += BK){
        float4 tmp =
            reinterpret_cast<float4 *>(&A[innerRowA * k + innerColA * 4])[0];
        shareA[(innerColA * 4 + 0) * BM + innerRowA] = tmp.x;
        shareA[(innerColA * 4 + 1) * BM + innerRowA] = tmp.y;
        shareA[(innerColA * 4 + 2) * BM + innerRowA] = tmp.z;
        shareA[(innerColA * 4 + 3) * BM + innerRowA] = tmp.w;
        reinterpret_cast<float4 *>(&shareB[innerRowB * BN + innerColB * 4])[0] =
            reinterpret_cast<float4 *>(&B[innerRowB * n + innerColB * 4])[0];

        __syncthreads();

        A += BK;
        B += BK*n; 

        for(uint dotIdx = 0; dotIdx < BK; dotIdx++){                             
            // load into register.
            // reinterpret_cast<float4 *>(&regA[0])[0] = 
            //     reinterpret_cast<float4 *>(&shareA[dotIdx*BM+threadRow])[0];
            // reinterpret_cast<float4 *>(&regA[4])[0] = 
            //     reinterpret_cast<float4 *>(&shareA[dotIdx*BM+threadRow+4])[0];
            // reinterpret_cast<float4 *>(&regB[0])[0] = 
            //     reinterpret_cast<float4 *>(&shareB[dotIdx*BM+threadRow])[0];
            // reinterpret_cast<float4 *>(&regB[4])[0] = 
            //     reinterpret_cast<float4 *>(&shareB[dotIdx*BM+threadRow+4])[0];
            for(uint i=0; i<TM; i++){
                regA[i] = shareA[dotIdx*BM+threadRow+i];
            }
            for(uint i=0; i<TN; i++){
                regB[i] = shareB[dotIdx*BM+threadCol+i];
            }
            for(uint resIdxM=0; resIdxM < TM; resIdxM++){
                for(uint resIdxN=0; resIdxN<TN; resIdxN++){
                    threadResults[resIdxM*TN+resIdxN] += regA[resIdxM]*regB[resIdxN];
                }
            }
        }
        __syncthreads();
    }

    for(uint resIdxM=0; resIdxM<TM; ++resIdxM){
        for(uint resIdxN=0; resIdxN < TN; resIdxN+=4){
            // load C vector into registers
            float4 tmp = reinterpret_cast<float4 *>(
                &C[(threadRow + resIdxM) * n + threadCol + resIdxN])[0];
            tmp.x = threadResults[resIdxM*TN + resIdxN];
            tmp.y = threadResults[resIdxM*TN + resIdxN+1];
            tmp.z = threadResults[resIdxM*TN + resIdxN+2];
            tmp.w = threadResults[resIdxM*TN + resIdxN+3];
            reinterpret_cast<float4 *>(
                &C[(threadRow + resIdxM) * m + threadCol + resIdxN])[0] = tmp;
        }
    }
   

}

void gemm_gpu_vectorized_memory(
    const int m, 
    const int n, 
    const int k,
    float *A,
    float alpha,
    float *B,
    float beta,
    float *C,
    hipblasHandle_t handle
){
    const int BM = 128;
    const int BN = 128;
    const int BK = 8;
    const int TM = 8;
    const int TN = 8;

    dim3 grid_dim = dim3(ceil(m/BM), ceil(n/BN));
    dim3 block_dim = dim3(BN*BM/(TM*TN));
    gemm_gpu_vectorized_mem_kernel<BM, BN, BK, TM, TN><<<grid_dim, block_dim>>>(m,n,k,A,alpha,B,beta,C);
}